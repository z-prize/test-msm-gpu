// Copyright Supranational LLC
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0

#include <hip/hip_runtime.h>
#include <iostream>
#include <nvToolsExt.h>
#include <string>
#include <sys/mman.h>

#include <ff/bls12-377.hpp>

#include <ec/jacobian_t.hpp>
#include <ec/xyzz_t.hpp>

typedef jacobian_t<fp_t> point_t;
typedef xyzz_t<fp_t> bucket_t;
typedef bucket_t::affine_inf_t affine_t;
typedef fr_t scalar_t;

#include <msm/pippenger.cuh>

#include "../bellman-cuda/src/bellman-cuda.h"

#ifndef __CUDA_ARCH__

static const size_t NUM_BATCH_THREADS = 2;
static thread_pool_t batch_pool(NUM_BATCH_THREADS);

typedef pippenger_t<bucket_t, point_t, affine_t, scalar_t> pipp_t;

struct bc_affine_t {
  fp_t x;
  fp_t y;
};

struct bc_proj_t {
  fp_t x;
  fp_t y;
  fp_t z;
};

// MSM context used store persistent state
struct Context {
  bc_mem_pool mem_pool;
  bc_affine_t *d_bases;
  bool *d_bases_inf_flags;
  scalar_t *h_scalars[2];
  scalar_t *h_results[2];
  scalar_t *d_scalars[2];
  scalar_t *d_results[2];
  bc_stream streams[2];
  bc_event events[2];
};

template<class bucket_t, class affine_t, class scalar_t>
struct RustContext {
  Context *context;
};

// not pretty, especially since bc_error was itself a cast from a hipError_t,
// but we're just trying to make our stuff work with the harness with minimal effort,
// not paint the sistine chapel
#define BC_CHK(ARG) \
{ \
  bc_error err = (ARG); \
  if (err != 0) { \
    hipError_t code = static_cast<hipError_t>(err); \
    return RustError(code); \
  } \
}

#define CUDA_CHK(ARG) \
{ \
  hipError_t code = (ARG); \
  if (code != hipSuccess) { \
    return RustError{code}; \
  } \
}

unsigned get_log_count(size_t npoints) {
  assert(npoints > 0);
  size_t tmp{npoints};
  unsigned log_count{0};
  while (tmp) {
    log_count++;
    tmp >>= 1;
  }
  return --log_count;
}

struct nvtxRangeGuard {
  nvtxRangeGuard(const char *label) { nvtxRangePush(label); }
  ~nvtxRangeGuard() { nvtxRangePop(); }
};

// Initialization function
// Allocate device storage, transfer bases
extern "C"
RustError mult_pippenger_init_bc(RustContext<bucket_t,
                                 affine_t, scalar_t> *context,
                                 const affine_t points[],
                                 const bool h_bases_inf_flags[],
                                 size_t npoints,
                                 size_t ffi_affine_sz)
{
  static_assert(sizeof(bc_proj_t) == sizeof(point_t));

  nvtxRangeGuard g{__FUNCTION__};

  // context is never deleted (leaks after timing loop in benches/msm.rs) but reference maintainer says that's ok.
  context->context = new Context{};
  auto *ctx = context->context;

  BC_CHK(msm_set_up());

  std::cout << "npoints " << npoints << std::endl;

  BC_CHK(bc_mem_pool_create(&ctx->mem_pool, 0));
  BC_CHK(bc_malloc((void**)&ctx->d_bases, sizeof(bc_affine_t) * npoints));
  BC_CHK(bc_malloc((void**)&ctx->d_bases_inf_flags, sizeof(bool) * npoints));

  for (int i = 0; i < 2; i++) {
    BC_CHK(bc_malloc_host((void**)&ctx->h_scalars[i], sizeof(scalar_t) * npoints));
    BC_CHK(bc_malloc_host((void**)&ctx->h_results[i], sizeof(bc_proj_t) * 256));
    BC_CHK(bc_malloc((void**)&ctx->d_scalars[i], sizeof(scalar_t) * npoints));
    BC_CHK(bc_malloc((void**)&ctx->d_results[i], sizeof(bc_proj_t) * 256));
    BC_CHK(bc_stream_create(&ctx->streams[i], false));
    BC_CHK(bc_event_create(&ctx->events[i], false, true));
  }

  // Copies bases from host to device.
  // Intent is to pick out limb data and skip inf flags, which are copied separately.
  CUDA_CHK(hipMemcpy2D(ctx->d_bases, sizeof(bc_affine_t), points, ffi_affine_sz,
                        sizeof(bc_affine_t), npoints, hipMemcpyHostToDevice));

  BC_CHK(bc_memcpy(ctx->d_bases_inf_flags, h_bases_inf_flags, sizeof(bool) * npoints));

  return RustError{hipSuccess};
}

// Peform MSM on a batch of scalars over fixed bases
extern "C"
RustError mult_pippenger_bc(RustContext<bucket_t,
                            affine_t, scalar_t> *context,
                            point_t* out,
                            const affine_t points[],
                            size_t npoints,
                            size_t batches,
                            const scalar_t scalars[],
                            size_t ffi_affine_sz)
{
  nvtxRangeGuard g{__FUNCTION__};

  auto *ctx = context->context;

  unsigned log_count = get_log_count(npoints);
  size_t bytes = sizeof(scalar_t) * npoints;

  for (unsigned batch = 0; batch < batches; batch++) {
    nvtxRangeGuard g{std::to_string(batch).c_str()};
    unsigned i = batch & 1; // alternate buffers + streams each iteration
    if (batch > 0) {
      CUDA_CHK(hipMemcpyAsync(ctx->h_scalars[i], scalars, bytes, hipMemcpyHostToHost,
                               static_cast<hipStream_t>(ctx->streams[i].handle)));
      CUDA_CHK(hipMemcpyAsync(ctx->d_scalars[i], ctx->h_scalars[i], bytes, hipMemcpyHostToDevice,
                               static_cast<hipStream_t>(ctx->streams[i].handle)));
    }
    msm_configuration cfg = {ctx->mem_pool,
                             ctx->streams[i],
                             ctx->d_bases,
                             batch == 0 ? (void*)scalars : (void*)ctx->d_scalars[i],
                             ctx->d_results[i],
                             log_count};
    cfg.bases_inf_flags = ctx->d_bases_inf_flags;
    BC_CHK(msm_execute_async(cfg));
    scalars += npoints; 
  }

  for (auto& stream : ctx->streams) {
    BC_CHK(bc_stream_synchronize(stream));
  }

  return RustError{hipSuccess};
}

#endif  //  __CUDA_ARCH__
